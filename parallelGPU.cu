﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <omp.h>

using namespace std;


__device__ char* XOR(char* a, char* b, int N)
{
	char* c = new char[N];
	for (int i = 0; i < N; i++)
	{
		if (a[i] != b[i])
			c[i] = '1';
		else
			c[i] = '0';
	}

	return c;
}


__device__ int weigth(char* a, int N)
{
	int cnt = 0;
	for (int i = 0; i < N; i++)
		if (a[i] == '1')
			cnt++;
	return cnt;
}



__global__ void kernel(char* A, int N, int K, int pow2k, int* gist, char* tmp, char* tmp_vec)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int mult, temp_i, w;
	
	if (id < pow2k)
	{
		mult = id % 2;
		temp_i = id / 2;
		if (mult)
			for (int z = 0; z < N; z++)
			{
				tmp[z] = A[z];
			}
		else
		{
			for (int z = 0; z < N; z++)
			{
				tmp[z] = '0';
			}
		}

		for (int j = 1; j < K; j++)
		{
			mult = temp_i % 2;
			temp_i /= 2;
			if (mult)
			{
				for (int z = 0; z < N; z++)
					tmp_vec[z] = A[N*j + z];
				tmp = XOR(tmp, tmp_vec, N);
			}
		}
		gist[id] = weigth(tmp, N);
	}
}


int main()
{
	ifstream fin;
	fin.open("in.txt");

	int N, K;
	fin >> N >> K;

	char* A = new char[N * K];

	for (int i = 0; i < N * K; i++)
		fin >> A[i];
	fin.close();

	int pow2k = pow(2, K);

	int threads = 32;
	int blocks = pow2k / threads + 1;

	int* gist = new int[pow2k];

	for (int i = 0; i < pow2k; i++)
		gist[i] = 0;

	double t1 = omp_get_wtime();

	char* A_d = new char [N * K];
	hipMalloc((void**)&A_d, N * K * sizeof(char));

	int* gist_d = new int  [pow2k];
	hipMalloc((void**)&gist_d, (pow2k) * sizeof(int));

	char* tmp = new char[N];
	char* tmp_vec = new char[N];

	char* tmp_d;
	hipMalloc((void**)&tmp_d, N * sizeof(char));
	char* tmp_vec_d;
	hipMalloc((void**)&tmp_vec_d, N * sizeof(char));

	hipMemcpy(A_d, A, N * K * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(gist_d, gist, (pow2k) * sizeof(int), hipMemcpyHostToDevice);

	kernel <<< blocks, threads >>> (A_d, N, K, pow2k, gist_d, tmp_d, tmp_vec_d);

	hipError_t cuerr;
	cuerr = hipGetLastError();
	if (cuerr != hipSuccess) {
		cout << "ERROR1!" << hipGetErrorString(cuerr) << endl;
	}

	int* gist_res = new int[pow2k];

	hipMemcpy(gist, gist_d, (pow2k) * sizeof(int), hipMemcpyDeviceToHost);

	cuerr = hipGetLastError();
	if (cuerr != hipSuccess) {
		cout << "ERROR2!" << hipGetErrorString(cuerr) << endl;
	}

	double t2 = omp_get_wtime();

	cout << t2 - t1 << endl;
	ofstream fout;
	fout.open("out.txt");

	int check = 0;

	int* res = new int[N + 1];

	for (int i = 0; i < N + 1; i++)
		res[i] = 0;

	for (int i = 0; i < pow2k; i++)
	{
		res[gist[i]] += 1;
	}

	for (int i = 0; i < N + 1; i++)
	{	if (res[i] != 0)
			fout << i << "\t" << res[i] << endl;
		check += res[i];
	}
	//cout << check;
	fout.close();
}

